#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include "tensorrt_llm/common/quantTypeUtils.cuh"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/rmsnormKernels.h"

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{

template <typename Tf, typename T>
__inline__ __device__ Tf compute_rmsnorm(Tf val, float s_variance, T const* gamma, T const* beta, int i)
{
    Tf ret = val * s_variance * cuda_cast<Tf>(gamma[i]);
    if (beta != nullptr)
    {
        ret = ret + cuda_cast<Tf>(beta[i]);
    }
    return ret;
}

/* Computes the rmsnorm https://pytorch.org/docs/stable/generated/torch.nn.rmsnorm.html
 * normed_output <- ( input / Sqrt(E[input²] + eps) ) * gamma + beta
 * input is [tokens, hidden_dim]. Mean and Variance are per-row (i.e. per-token)
 *
 * One CTA handles one row.
 *
 *
 * USE_SHMEM controls if we cache input values into shared memory
 *
 * Optional: with dynamic scaling, the last pass doesn't write immediately but finds the
 *           amax per row. A final pass scales to int8 accordingly, and writes output to
 *           normed_output_quant.
 */
template <typename T, typename QuantT, bool USE_SHMEM>
__global__ void generalRmsNorm(T const* input, T const* gamma, T const* beta, T* normed_output, float const eps,
    int tokens, int hidden_dim, float const* clampPtr, float const* scale_orig_quant_per_tensor,
    float* scale_orig_quant_per_token, float* sum_per_token, QuantT* normed_output_quant, bool hasFp8MinScaling)
{
    constexpr auto num_elems_T = num_elems<T>::value;
    // using int8_packed_t = typename packed_as<int8_t, num_elems_T>::type;
    // using fp8_packed_t = typename packed_as<__hip_fp8_e4m3_fnuz, num_elems_T>::type;
    using QuantT_packed_t = typename packed_as<QuantT, num_elems_T>::type;
    using float_packed_t = typename packed_as<float, num_elems_T>::type;
    using T_scalar = typename packed_as<T, 1>::type;

    // The clamping minimum / maximum values.
    T const clampMin = cuda_cast<T>(clampPtr ? clampPtr[0] : -FLT_MAX);
    T const clampMax = cuda_cast<T>(clampPtr ? clampPtr[1] : FLT_MAX);

    // The quantized data type's maximum value (upper-bound).
    static constexpr float MAX_QUANT_VAL = QuantTypeStaticVals<QuantT>::MAX_VAL;
    // The minimum scaling factor (lower-bound).
    static constexpr float MIN_SCALING_FACTOR = QuantTypeStaticVals<QuantT>::MIN_SCALING_FACTOR;
    static constexpr float MIN_SCALING_FACTOR_RCP = QuantTypeStaticVals<QuantT>::MIN_SCALING_FACTOR_RCP;

    extern __shared__ __align__(sizeof(float)) char _shmem[];
    T* shmem = reinterpret_cast<T*>(_shmem);

    __shared__ float s_variance;

    int const tidx = threadIdx.x;
    int const bidx = blockIdx.x;

    float variance = 0.0f;
    float local_var_sum = 0.0f;

    int const n_elems = hidden_dim / num_elems_T;
    for (int i = tidx; i < n_elems; i += blockDim.x)
    {
        T const val = input[bidx * n_elems + i];
        if (USE_SHMEM)
        {
            shmem[i] = val;
        }

        float_packed_t const val_f = cuda_cast<float_packed_t>(val);

        local_var_sum += cuda_sum<float>(val_f * val_f);
    }

    float packed[1] = {local_var_sum};
    blockReduceSumV2<float, 1>(packed);
    variance = packed[0];

    if (threadIdx.x == 0)
    {
        variance = (variance / hidden_dim); // Var[x] = E[x²]
        s_variance = rsqrtf(variance + eps);
    }
    __syncthreads();

    bool const with_per_token_scaling = scale_orig_quant_per_token != nullptr;
    bool const with_per_tensor_scaling = scale_orig_quant_per_tensor != nullptr;
    bool const with_per_token_sum = sum_per_token != nullptr;

    float_packed_t const scale_orig_quant
        = cuda_cast<float_packed_t>(with_per_tensor_scaling ? *scale_orig_quant_per_tensor : 0.0f);
    T_scalar amax = 1e-6f;
    float local_sum = 0.f;

    for (int i = tidx; i < n_elems; i += blockDim.x)
    {
        int const index = bidx * n_elems + i;
        float_packed_t const val_f = cuda_cast<float_packed_t>(USE_SHMEM ? shmem[i] : input[index]);
        T val = cuda_cast<T>(compute_rmsnorm(val_f, s_variance, gamma, beta, i));

        if (with_per_token_scaling)
        {
            val = cuda_clamp(val, clampMin, clampMax);
            amax = cuda_max(cuda_max<T_scalar, T>(cuda_abs(val)), amax);
            if (USE_SHMEM)
            {
                shmem[i] = val;
            }
        }
        else if (with_per_tensor_scaling)
        {
            val = cuda_clamp(val, clampMin, clampMax);
            reinterpret_cast<QuantT_packed_t*>(normed_output_quant)[index]
                = cuda_cast<QuantT_packed_t>(cuda_cast<float_packed_t>(val) * scale_orig_quant);
        }
        else
        {
            normed_output[index] = val;
        }

        if (with_per_token_sum)
        {
            local_sum += cuda_sum<float>(cuda_cast<float_packed_t>(val));
        }
    }

    if (with_per_token_scaling)
    {
        float abs_max_f = blockAllReduceMax(cuda_cast<float>(amax));
        float const dynamic_per_token_scale
            = hasFp8MinScaling ? fminf(MAX_QUANT_VAL / abs_max_f, MIN_SCALING_FACTOR_RCP) : (MAX_QUANT_VAL / abs_max_f);
        for (int i = tidx; i < n_elems; i += blockDim.x)
        {
            int const index = bidx * n_elems + i;
            float_packed_t val_f = cuda_cast<float_packed_t>(USE_SHMEM ? shmem[i] : input[index]);
            if (!USE_SHMEM)
            {
                val_f = compute_rmsnorm(val_f, s_variance, gamma, beta, i);
            }

            reinterpret_cast<QuantT_packed_t*>(normed_output_quant)[index]
                = cuda_cast<QuantT_packed_t>(val_f * cuda_cast<float_packed_t>(dynamic_per_token_scale));
        }
        if (tidx == 0)
        {
            scale_orig_quant_per_token[bidx] = hasFp8MinScaling
                ? cuda_max(abs_max_f / MAX_QUANT_VAL, MIN_SCALING_FACTOR)
                : abs_max_f / MAX_QUANT_VAL;
        }
    }

    if (with_per_token_sum)
    {
        float packed_sum[1] = {local_sum};
        blockReduceSumV2<float, 1>(packed_sum);
        if (tidx == 0)
        {
            sum_per_token[bidx] = packed_sum[0];
        }
    }
}

template <typename T, typename QuantT>
void dispatch_rmsnorm_type_square_method(T const* input, T const* gamma, T const* beta, T* normed_output,
    float const eps, int tokens, int hidden_dim, float const* clampPtr, float const* scale_orig_quant_per_tensor,
    float* scale_orig_quant_per_token, float* sum_per_token, QuantT* normed_output_quant, bool const hasFp8MinScaling,
    dim3 const grid, dim3 const block, size_t const shmem_size, hipStream_t stream)
{
    // Do we use shared memory to cache intermediate results.
    bool use_shmem = true;
    if (shmem_size >= (48 << 10))
    {
        hipError_t ret = hipFuncSetAttribute(reinterpret_cast<const void*>(
            generalRmsNorm<T), QuantT, true>, hipFuncAttributeMaxDynamicSharedMemorySize, shmem_size);
        // Use shared memory when the capactity is enough.
        use_shmem = (ret == hipSuccess);
    }

    if (use_shmem)
    {
        generalRmsNorm<T, QuantT, true><<<grid, block, shmem_size, stream>>>(input, gamma, beta, normed_output, eps,
            tokens, hidden_dim, clampPtr, scale_orig_quant_per_tensor, scale_orig_quant_per_token, sum_per_token,
            normed_output_quant, hasFp8MinScaling);
    }
    else
    {
        generalRmsNorm<T, QuantT, false><<<grid, block, shmem_size, stream>>>(input, gamma, beta, normed_output, eps,
            tokens, hidden_dim, clampPtr, scale_orig_quant_per_tensor, scale_orig_quant_per_token, sum_per_token,
            normed_output_quant, hasFp8MinScaling);
    }
}

template <typename T, typename QuantT>
void dispatch_rmsnorm_type(T const* input, T const* gamma, T const* beta, T* normed_output, float const eps, int tokens,
    int hidden_dim, float const* clampPtr, float const* scale_orig_quant_per_tensor, float* scale_orig_quant_per_token,
    float* sum_per_token, QuantT* normed_output_quant, bool const hasFp8MinScaling, dim3 const grid, dim3 const block,
    size_t const shmem_size, hipStream_t stream)
{
    dispatch_rmsnorm_type_square_method(input, gamma, beta, normed_output, eps, tokens, hidden_dim, clampPtr,
        scale_orig_quant_per_tensor, scale_orig_quant_per_token, sum_per_token, normed_output_quant, hasFp8MinScaling,
        grid, block, shmem_size, stream);
}

template <typename T, typename QuantT>
void invokeGeneralRmsNorm(T* out, T const* input, T const* gamma, T const* beta, float const eps, int const tokens,
    int const hidden_dim, QuantMode quantMode, hipStream_t stream, float const* clampPtr, float const* scale,
    float* dynamic_scale, float* sum_per_token, QuantT* normed_output_quant)
{
    dim3 grid(tokens);
    dim3 block(min(hidden_dim, 1024));
    // Make sure block.x is multiple of 32 for warp shuffle to work
    block.x = 32 * ((block.x + 31) / 32);

    constexpr size_t vec_size = 2;
    size_t const shmem_size = hidden_dim * sizeof(T);
    bool const use_vec_type = (hidden_dim % vec_size == 0)
        && (std::is_same<T, half>::value
#ifdef ENABLE_BF16
            || std::is_same<T, __hip_bfloat16>::value
#endif
        );

    // Enable min_scaling_factor if it is fp8 rowwise per-token quantization.
    bool hasFp8MinScaling = quantMode.hasFp8RowWise();

    if (use_vec_type)
    {
        using Tp = typename packed_as<T, vec_size>::type;
        dispatch_rmsnorm_type(reinterpret_cast<Tp const*>(input), reinterpret_cast<Tp const*>(gamma),
            reinterpret_cast<Tp const*>(beta), reinterpret_cast<Tp*>(out), eps, tokens, hidden_dim, clampPtr, scale,
            dynamic_scale, sum_per_token, normed_output_quant, hasFp8MinScaling, grid, block, shmem_size, stream);
    }
    else
    {
        dispatch_rmsnorm_type(input, gamma, beta, out, eps, tokens, hidden_dim, clampPtr, scale, dynamic_scale,
            sum_per_token, normed_output_quant, hasFp8MinScaling, grid, block, shmem_size, stream);
    }
}

#define INSTANTIATE_GENERAL_RMSNORM(T, QuantT)                                                                         \
    template void invokeGeneralRmsNorm(T* out, const T* input, const T* gamma, const T* beta, const float eps,         \
        const int tokens, const int hidden_dim, QuantMode quantMode, hipStream_t stream, float const* clampPtr,       \
        const float* scale, float* dynamic_scale, float* sum_per_token, QuantT* normed_output_quant);

INSTANTIATE_GENERAL_RMSNORM(float, int8_t);
INSTANTIATE_GENERAL_RMSNORM(half, int8_t);

#ifdef ENABLE_BF16
INSTANTIATE_GENERAL_RMSNORM(__hip_bfloat16, int8_t);
#endif

#ifdef ENABLE_FP8
INSTANTIATE_GENERAL_RMSNORM(float, __hip_fp8_e4m3_fnuz);
INSTANTIATE_GENERAL_RMSNORM(half, __hip_fp8_e4m3_fnuz);
#ifdef ENABLE_BF16
INSTANTIATE_GENERAL_RMSNORM(__hip_bfloat16, __hip_fp8_e4m3_fnuz);
#endif
#endif

} // namespace kernels
} // namespace tensorrt_llm
